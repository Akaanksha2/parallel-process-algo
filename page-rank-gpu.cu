#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DAMPING_FACTOR 0.85f

__global__ void pagerank_kernel(float *d_A, float *d_R, float *d_sum, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        float sum = 0;
        for (int j = 0; j < N; j++) {
            sum += d_A[idx * N + j] * d_R[j];
        }
        d_sum[idx] = sum;
        d_R[idx] = (1 - DAMPING_FACTOR) / N + DAMPING_FACTOR * sum;
    }
}

int main(int argc, char **argv) {
    if (argc != 2) {
        fprintf(stderr, "Usage: %s input_file\n", argv[0]);
        exit(EXIT_FAILURE);
    }
    char *input_file = argv[1];

    // Open file
    FILE *f = fopen(input_file, "r");
    if (f == NULL) {
        fprintf(stderr, "Failed to open file %s\n", input_file);
        exit(EXIT_FAILURE);
    }

    // Read matrix size
    int N;
    fscanf(f, "%d", &N);

    // Allocate host memory
    float *A = (float *) malloc(N * N * sizeof(float));
    float *R = (float *) malloc(N * sizeof(float));
    float *sum = (float *) malloc(N * sizeof(float));

    // Read matrix from file
    for (int i = 0; i < N * N; i++) {
        fscanf(f, "%f", &A[i]);
	//printf("Read A[%d] = %f\n", i, A[i]);	
    }

    // Initialize R to 1 / N
    for (int i = 0; i < N; i++) {
        R[i] = 1.0f / N;
    }

    // Allocate device memory
    float *d_A, *d_R, *d_sum;
    hipMalloc((void **) &d_A, N * N * sizeof(float));
    hipMalloc((void **) &d_R, N * sizeof(float));
    hipMalloc((void **) &d_sum, N * sizeof(float));

    // Copy input to device memory
    hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_R, R, N * sizeof(float), hipMemcpyHostToDevice);

    // Run kernel for 100 iterations
    for (int i = 0; i < 100; i++) {
        pagerank_kernel<<<(N + 255) / 256, 256>>>(d_A, d_R, d_sum, N);
        hipMemcpy(R, d_R, N * sizeof(float), hipMemcpyDeviceToHost);
    }

    // Print result
    for (int i = 0; i < N; i++) {
        printf("R[%d] = %f\n", i, R[i]);
    }

    // Free memory
    free(A);
    free(R);
    free(sum);
    hipFree(d_A);
    hipFree(d_R);
    hipFree(d_sum);

    return 0;
}
